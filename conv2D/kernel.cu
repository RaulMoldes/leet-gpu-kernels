#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>


__device__ inline int get_global_idx() {
    return blockIdx.x * blockDim.x + threadIdx.x;
}

__device__ inline int get_global_idy() {
    return blockIdx.y * blockDim.y + threadIdx.y;
}



extern "C" __global__ void conv2d_forward(
    const float* input,
    const float* filter,
    const float* bias,
    float* output,
    int batch_size,
    int in_channels,
    int in_height,
    int in_width,
    int out_channels,
    int out_height,
    int out_width,
    int kernel_height,
    int kernel_width,
    int stride_h,
    int stride_w,
    int pad_h,
    int pad_w
) {
    const int TILE_SIZE = 16;

    // Use shared memory for input and filter
    // Shared memory size: input tile + filter
    // Total shared memory size: (TILE_SIZE + kernel_height - 1) *
    extern __shared__ float shared_mem[];

    // Compute the size of the input tile
    // Input tile size is the size of the output tile plus the kernel size minus 1
    // This accounts for the padding needed for the convolution
    int input_tile_h = TILE_SIZE + kernel_height - 1;
    int input_tile_w = TILE_SIZE + kernel_width - 1;
    int filter_size = kernel_height * kernel_width;

    // Partition shared memory
    // First part for input tile, second part for filter
    // shared_mem size: input_tile_h * input_tile_w + filter_size
    float* shared_input = shared_mem;
    float* shared_filter = shared_mem + input_tile_h * input_tile_w;

    int out_x = blockIdx.x * TILE_SIZE + threadIdx.x;
    int out_y = blockIdx.y * TILE_SIZE + threadIdx.y;
    int out_c = blockIdx.z % out_channels;
    int batch_idx = blockIdx.z / out_channels;

    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int thread_id = ty * blockDim.x + tx;
    int threads_per_block = blockDim.x * blockDim.y;

    // Compute the output coordinates
    bool valid_output = (out_x < out_width && out_y < out_height &&
        out_c < out_channels && batch_idx < batch_size);

    float result = 0.0f;

    // Process each input channel
    for (int in_c = 0; in_c < in_channels; in_c++) {

        // 1. Load the filter cooperatively into shared memory
        // Each thread loads one element of the filter
        for (int i = thread_id; i < filter_size; i += threads_per_block) {
            int ky = i / kernel_width;
            int kx = i % kernel_width;

            int filter_idx = out_c * (in_channels * kernel_height * kernel_width) +
                in_c * (kernel_height * kernel_width) +
                ky * kernel_width + kx;
            shared_filter[i] = filter[filter_idx];
        }

        // 2. Load the input tile into shared memory
        // Each thread loads one element of the input tile
        int input_elements = input_tile_h * input_tile_w;
        for (int i = thread_id; i < input_elements; i += threads_per_block) {
            int tile_y = i / input_tile_w;
            int tile_x = i % input_tile_w;

            // Calculate the input coordinates based on the tile position
            int in_x = blockIdx.x * TILE_SIZE * stride_w - pad_w + tile_x;
            int in_y = blockIdx.y * TILE_SIZE * stride_h - pad_h + tile_y;

            if (in_x >= 0 && in_x < in_width && in_y >= 0 && in_y < in_height) {
                int input_idx = batch_idx * (in_channels * in_height * in_width) +
                    in_c * (in_height * in_width) +
                    in_y * in_width + in_x;
                shared_input[i] = input[input_idx];
            }
            else {
                shared_input[i] = 0.0f; // Zero padding
            }
        }

        __syncthreads();

        // 3. Perform the convolution operation
        // Each thread computes a part of the output
        if (valid_output) {
            for (int ky = 0; ky < kernel_height; ky++) {
                for (int kx = 0; kx < kernel_width; kx++) {
                    // Position in the shared memory tile
                    int shared_y = ty * stride_h + ky;
                    int shared_x = tx * stride_w + kx;

                    // Verify if the shared memory indices are within bounds
                    if (shared_y < input_tile_h && shared_x < input_tile_w) {
                        int input_idx = shared_y * input_tile_w + shared_x;
                        int filter_idx = ky * kernel_width + kx;

                        result += shared_input[input_idx] * shared_filter[filter_idx];
                    }
                }
            }
        }

        __syncthreads();
    }

    // Add bias if provided
    // Bias is added only if the output is valid
    if (valid_output) {
        if (bias != nullptr) {
            result += bias[out_c];
        }

        int output_idx = batch_idx * (out_channels * out_height * out_width) +
            out_c * (out_height * out_width) +
            out_y * out_width + out_x;
        output[output_idx] = result;
    }

}


extern "C" void solve(const float* input, const float* kernel, float* output,
    int input_rows, int input_cols,
    int kernel_rows, int kernel_cols) {
    // Parameters
    int batch_size = 1;
    int in_channels = 1;
    int out_channels = 1;
    int stride_h = 1;
    int stride_w = 1;
    int pad_h = 0;
    int pad_w = 0;

    int in_height = input_rows;
    int in_width = input_cols;
    int out_height = (in_height + 2 * pad_h - kernel_rows) / stride_h + 1;
    int out_width = (in_width + 2 * pad_w - kernel_cols) / stride_w + 1;

    // Sizes
    size_t input_size = batch_size * in_channels * in_height * in_width * sizeof(float);
    size_t filter_size = out_channels * in_channels * kernel_rows * kernel_cols * sizeof(float);
    size_t output_size = batch_size * out_channels * out_height * out_width * sizeof(float);
    size_t bias_size = out_channels * sizeof(float);

    // Allocate memory
    float* d_input, * d_filter, * d_output, * d_bias;
    hipMalloc(&d_input, input_size);
    hipMalloc(&d_filter, filter_size);
    hipMalloc(&d_output, output_size);
    hipMalloc(&d_bias, bias_size);
    hipMemset(d_bias, 0, bias_size);  // Use zero bias for now

    // Copy input & kernel
    hipMemcpy(d_input, input, input_size, hipMemcpyHostToDevice);
    hipMemcpy(d_filter, kernel, filter_size, hipMemcpyHostToDevice);

    // Kernel configuration
    const int TILE_SIZE = 16;
    dim3 blockDim(TILE_SIZE, TILE_SIZE);
    dim3 gridDim((out_width + TILE_SIZE - 1) / TILE_SIZE,
        (out_height + TILE_SIZE - 1) / TILE_SIZE,
        batch_size * out_channels);

    int input_tile_h = TILE_SIZE + kernel_rows - 1;
    int input_tile_w = TILE_SIZE + kernel_cols - 1;
    int shared_mem_size = (input_tile_h * input_tile_w + kernel_rows * kernel_cols) * sizeof(float);

    // Launch kernel
    conv2d_forward << <gridDim, blockDim, shared_mem_size >> > (
        d_input, d_filter, d_bias, d_output,
        batch_size, in_channels,
        in_height, in_width,
        out_channels, out_height, out_width,
        kernel_rows, kernel_cols,
        stride_h, stride_w,
        pad_h, pad_w
        );

    // Copy result back
    hipMemcpy(output, d_output, output_size, hipMemcpyDeviceToHost);

    // Cleanup
    hipFree(d_input);
    hipFree(d_filter);
    hipFree(d_output);
    hipFree(d_bias);
}
